#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <vector>
using namespace std;

#include "Timer.hpp"
#include "cudaData.cuh"

__global__ void cudacopy(float* b, float* a, int size){
    // 乾式寫法
    const int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(idx<size){
        b[idx]=a[idx];
    }
    // 迴圈寫法
    for(int i=threadIdx.x; i<size; i+=blockDim.x){
        b[i]=a[i];
    }
}
void cpucopoy(float* b, float* a, int size) {
    for(int i=0; i<size; ++i){
        b[i]=a[i];
    }
}
void testCuda(size_t size) {
    Timer T;

    // 配置主機記憶體
    vector<float> img_data(size), cpu_data(size), gpu_data(size);
    float* a = img_data.data(); // 原始資料
    float* b = cpu_data.data(); // CPU計算後資料
    float* c = gpu_data.data(); // GPU輸出回來資料
    // 設置初值
    float test_val=7;
    for(int i=0; i<size; i++){
        a[i]=test_val;
    }

    // 配置顯示記憶體, 載入資料.
    T.start();
    CudaData<float> gpuDataIn(a, size), gpuDataOut(size);
    gpuDataOut.memset(0, size);
    T.print("  Cuda Data malloc and copy");

    // 網格區塊設定. (與 kernel for 的次數有關)
    const size_t blkDim=16;
    int grid(size/blkDim+1);  // 網格要含蓋所有範圍, 所以除完要加 1.
    int block(blkDim);        // 區塊設定 16x16.

    // Cuda Kernel 執行運算
    T.start();
    cudacopy<<<grid,block>>>(gpuDataOut, gpuDataIn, size);
    T.print("  Cuda-copy");
    // 取出GPU資料
    gpuDataOut.memcpyOut(c, size);

    // CPU 執行運算
    T.start();
    cpucopoy(b, a, size);
    T.print("  Cpu-copy");

    // 測試
    bool f=0;
    for(size_t i = 0; i < size; i++) {
        if(c[i] != b[i]) {f=1;}
        cout << a[i] << "| ";
        cout << c[i] << ", ";
        cout << b[i] << endl;
    }

    // 測試報告
    if(f==0) {
        cout << "test ok" << endl;
    } else {
        cout << "test Error" << endl;
    }
}

int main(){
    Timer T; 
    testCuda(10);
    T.print("ALL time.");

    system("pause");
    return 0;
}